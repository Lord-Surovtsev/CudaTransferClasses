#include "hip/hip_runtime.h"
#include "stdio.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

//#include "utils.h"

#include <iostream>
using namespace std;


#ifndef CHECKCUDAERRORS_H
#define CHECKCUDAERRORS_H

#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include "Point3D.h"

int CudaCheckLastError()
{
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        fflush(stdout);
        return 1;
        //exit(-1);
    }
    printf("no errors\n");
    fflush(stdout);
    return 0;
}

#endif // CHECKCUDAERRORS_H


__global__
void operationKernel(Point3D* d_points, const int count)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if (count <= myId)
    {
        return;
    }
    d_points[myId]._y = myId;
}


void CallKernel(Point3D* h_points, const int count)
{
    const int threads = 16;
    const dim3 gridSize((count + threads - 1) / threads);
    const dim3 blockSize(threads);

    const size_t memLen = sizeof(Point3D) * count;
    Point3D* d_points;
    hipMalloc((void**)&d_points, memLen );
    hipMemcpy(d_points, h_points, memLen, hipMemcpyHostToDevice);
    CudaCheckLastError();

    operationKernel<<<gridSize, blockSize>>>(d_points, count);

    hipDeviceSynchronize();

    hipMemcpy(h_points, d_points, memLen, hipMemcpyDeviceToHost);
    hipFree(d_points);
}
